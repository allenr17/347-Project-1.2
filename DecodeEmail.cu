
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
using namespace std;


__global__ void decode(char *msg);

int main(int argc, char *argv[]){

    if (argc > 2){
        cout << "You have entered too many arguments, the program will now exit.\n";
        exit(0);
    }else if(argc == 1){
        cout << "You have entered too few arguments, the program will now exit.\n";
        exit(0);
    }

    char* filename = argv[1];

    cout << "File name: " << filename << '\n';

    char *r;
    char *dev_r;
    r = (char*)malloc(sizeof(char) * (256));
    hipMalloc((void**)&dev_r, sizeof(char) * (256));

    cout << r << '\n';

    FILE *file;
    file = fopen(filename, "r");

    if (file){

        cout << "File opened" << '\n';

        fscanf(file, "%s,", r);

        cout << "File scanned\n";
        
    }else{
        cout << "That file does not exist, the program will now exit.\n";
        exit(0);
    }
    
    cout << "Original Text:\n" << r << "\n\n";

    hipMemcpy(dev_r, r, sizeof(char) * (256), hipMemcpyHostToDevice);

    decode<<<1, (sizeof(char) * (256))>>>(dev_r);

    hipDeviceSynchronize();

    hipMemcpy(r, dev_r, sizeof(char) * (256), hipMemcpyDeviceToHost);

    cout << "Decoded Text:\n" << r << '\n';

    hipFree(dev_r);
    free(r);

    exit(0);

}

__global__ void decode(char *m){
    int i = threadIdx.x;
    int temp = -1;
    if (int(m[i]) != 0){
        temp = int(m[i]);
        temp -= 1;
        m[i] = char(temp);
    }

}


